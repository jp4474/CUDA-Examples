#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

#define N 102400

void add_1d(int *a, int *b, int *c) {
    int n = sizeof(a) / sizeof(int);
    for(int i = 0; i < n; i++){
        c[i] = a[i] + b[i];
    }
}

__global__ void add_1d_kernel(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // If the index is within the range of the array, add the corresponding elements from arrays 'a' and 'b' and store the result in array 'c'.
    if (index < n){
        c[index] = a[index] + b[index];
    }
}

void add_2d(int *a, int *b, int *c, int width, int height) {
    for(int i = 0; i < height; i++){
        for(int j = 0; j < width; j++){
            c[i*width + j] = a[i*width + j] + b[i*width + j];
        }
    }
}

__global__ void add_2d_kernel(int *a, int *b, int *c, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        int index = y * width + x;
        c[index] = a[index] + b[index];
    }
}

void add_3d(int *a, int *b, int *c, int width, int height, int depth) {
    for(int i = 0; i < depth; i++){
        for(int j = 0; j < height; j++){
            for(int k = 0; k < width; k++){
                c[i*width*height + j*width + k] = a[i*width*height + j*width + k] + b[i*width*height + j*width + k];
            }
        }
    }
}


__global__ void add_3d_kernel(int *a, int *b, int *c, int width, int height, int depth) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x < width && y < height && z < depth) {
        int index = z * width * height + y * width + x;
        c[index] = a[index] + b[index];
    }
}

int main() {

    // 1D
    // initialize matrices a and b 
    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];
    
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i+1;
    }
    auto cpu_start = std::chrono::high_resolution_clock::now();

    add_1d(a, b, c);

    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_elapsed = cpu_end - cpu_start;

    std::cout << "Elapsed time: " << cpu_elapsed.count() << " seconds.\n";

    auto gpu_start = std::chrono::high_resolution_clock::now();
    // Allocate memory on the GPU
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    // Copy data from the host to the device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Call the kernel function
    dim3 threadsPerBlock(1024);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    add_1d_kernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    int *result = new int[N];

    // Copy data from the device to the host
    hipMemcpy(result, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    auto gpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_elapsed = gpu_end - gpu_start;

    std::cout << "Elapsed time: " << gpu_elapsed.count() << " seconds.\n";

    for(int i = 0; i < N; i++) {
        std::cout << c[i] << " " << result[i] << "\n";
        assert(c[i]==result[i]);
    }

    std::cout << "1D test passed.\n";
}